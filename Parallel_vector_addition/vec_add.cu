
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
  *c = *a + *b;
}

int main() {
	// Initialize data on host
    int a = 2, b = 7; 
	int c; 
	int size = sizeof(int);

	// Allocate memory on device
	int *d_a, *d_b, *d_c;
	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	// Copy data from host to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Do the calculation on the device
    add<<<1,1>>>(d_a, d_b, d_c);

	// Copy the results from the device to the host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "The result is : " << c << '\n';
}
