
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int* a, int* b, int* c, int n) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index < n) {
	c[index] = a[index] + b[index];
  }
}

#define N 512
#define M 128

void random_ints(int* a) {
  for (int i{}; i < N; ++i) {
	a[i] = rand();
  }
}

int main() {
  // Allocate memory on host
  int *a, *b, *c;
  int size{sizeof(int) * N};
  a = (int*)malloc(size);
  b = (int*)malloc(size);
  c = (int*)malloc(size);
  // Initialize input
  random_ints(a);
  random_ints(b);

  // Allocate memory on device
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Copy data from host to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  // Calculate the output on device
  add<<<N/M, M>>>(d_a, d_b, d_c, N);

  // Copy the output back to host
  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
  hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Print the output
  int sum{};
  for(int i{}; i < size; ++i) {
	sum += c[i];
  }
  std::cout << "The sum is : " << sum << '\n';

  // Free memory
  free(a);
  free(b);
  free(c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
