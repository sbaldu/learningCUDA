
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) { 
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; 
}

#define N 32

void random_ints(int *a) {
  for (int i{}; i < N; ++i) {
    a[i] = rand();
  }
}

int main() {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Alloc memory on host and initialize data
  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  random_ints(a);
  random_ints(b);

  // Alloc memory on device
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Copy data to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Calculate the output on the device
  add<<<N, 1>>>(a, b, c);

  // Copy the output to the host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  int sum{};
  for (int i{}; i < N; ++i) {
    sum += c[i];
  }

  // Free memory on host
  free(a);
  free(b);
  free(c);
  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "The sum is : " << sum << '\n';
}
