
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <random>
#include <vector>

__global__ void reverse(const int* in, int* out, int n) {
  unsigned int th{threadIdx.x};

  // Declare array on shared memory
  extern __shared__ int temp[];

  // Load data in the shared memory array
  temp[th] = in[th];

  // Make sure that all the threads are done building the shared array
  __syncthreads();

  if (th < n) {
    out[th] = temp[n - th - 1];
  }
}
__host__ void initialize(std::vector<int>& vec) {
  for (int i{}; i < vec.size(); ++i) {
    vec[i] = std::rand();
  }
}

__host__ void verify(std::vector<int> const& input, std::vector<int> const& output) {
  size_t n{input.size()};
  for (int i{}; i < n; ++i) {
    assert(input[i] == output[n - i - 1]);
  }

  std::cout << "The result is correct!\n";
}

int main() {
  const int threadsPerBlock{256};
  const int N{threadsPerBlock};
  const int size{N * sizeof(int)};

  std::vector<int> in(N), out(N);
  initialize(in);

  // Allocate memory on device
  int *d_in, *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);

  // Copy memory to device, execute kernel and copy result back to host
  hipMemcpy(d_in, in.data(), size, hipMemcpyHostToDevice);
  reverse<<<1, threadsPerBlock, N * sizeof(int)>>>(d_in, d_out, N);
  hipMemcpy(out.data(), d_out, size, hipMemcpyDeviceToHost);

  // Verify result
  verify(in, out);

  // Free memory
  hipFree(d_in);
  hipFree(d_out);
}
