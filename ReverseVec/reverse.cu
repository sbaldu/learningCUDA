
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <vector>

#define n 10

__global__ void reverse(int* in, int* out) {
  int index = threadIdx.x;
  out[index] = in[n - index - 1];
}

int main() {
  // Allocate memory on host
  std::vector<int> a(n);
  std::vector<int> b(n);
  std::iota(a.begin(), a.end(), 0);
  std::iota(b.begin(), b.end(), 0);

  int size{sizeof(int) * n};

  // Allocate memory on device
  int *d_a, *d_b;
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);

  // Move data from host to device
  hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);

  // Calculate output on device
  reverse<<<1, n>>>(d_a, d_b);

  // Move output from device to host
  hipMemcpy(b.data(), d_b, size, hipMemcpyDeviceToHost);

  // Print output
  for (auto const& x : b) {
    std::cout << x << '\n';
  }

  // Free memory
  hipFree(d_a);
  hipFree(d_b);
}
