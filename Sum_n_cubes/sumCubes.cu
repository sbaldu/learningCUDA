
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <numeric>
#include <vector>

#define N 100

__global__ void cube(int* a, int* b) {
  b[threadIdx.x] = std::pow(a[threadIdx.x], 3);
}

int main() {
  // Allocate memory on host
  std::vector<int> in(N);
  std::vector<int> out(N);
  std::iota(in.begin(), in.end(), 1);

  int size{sizeof(int) * N};

  // Allocate memory on device
  int* d_in;
  int* d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);

  // Copy memory to device
  hipMemcpy(d_in, in.data(), size, hipMemcpyHostToDevice);

  // Calculate the cubes and save them on the output pointer
  cube<<<1, N>>>(d_in, d_out);

  // Copy the output back to host
  hipMemcpy(out.data(), d_out, size, hipMemcpyDeviceToHost);

  // Calculate the sum
  int sum{};
  for(auto const& x : out) {
	sum += x;
  }
  std::cout << "The final output is : " << sum << '\n';

  // Free memory
  hipFree(d_in);
  hipFree(d_out);
}
